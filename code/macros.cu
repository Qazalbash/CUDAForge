
#include <hip/hip_runtime.h>
// Copied from Book: Professional CUDA-C Programming
#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess) {                                            \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                           \
        }                                                                      \
    }

#define DEVICE_INFO()                                          \
    {                                                          \
        int            dev = 0;                                \
        hipDeviceProp_t deviceProp;                             \
        CHECK(hipGetDeviceProperties(&deviceProp, dev));      \
        printf("Using Device %d: %s\n", dev, deviceProp.name); \
        CHECK(hipSetDevice(dev));                             \
    }