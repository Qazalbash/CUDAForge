#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char** argv) {
    const int nElem = 1024;

    dim3 block(1024);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 512;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 256;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    block.x = 128;
    grid.x  = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d\n", grid.x, block.x);

    hipDeviceReset();

    return EXIT_SUCCESS;
}